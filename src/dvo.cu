#include "hip/hip_runtime.h"
// Copyright 2016 Robert Maier, Technical University Munich
#include "dvo.hpp"
#include "helper.h"

#include <iostream>
#include <sstream>
#include <string>
#include <iomanip>
#include <stdio.h>
#include <ctime>

#include "helper.h"

#include <Eigen/Cholesky>
#include <sophus/se3.hpp>

#include <opencv2/imgproc/imgproc.hpp>
#include <opencv2/highgui/highgui.hpp>

#include <hip/hip_runtime.h>
#include <thrust/count.h>
#include <hipblas.h>

#include <math.h>
#include <thrust/execution_policy.h>



DVO::DVO() :
    numPyramidLevels_(5),
    useWeights_(true),
    numIterations_(500),
    algo_(GaussNewton)
{
	for(int i=0;i<NUM_STREAMS;i++)
		hipStreamCreate(&streams[i]);
}


DVO::~DVO()
{
    for (int i = 0; i < numPyramidLevels_; ++i)
    {
        hipFree(d_J_[i]); CUDA_CHECK;
        hipFree(d_residuals_[i]);CUDA_CHECK;
        hipFree(d_weights_[i]);CUDA_CHECK;
    }
	for(int i=0;i<NUM_STREAMS;i++)
		hipStreamDestroy(streams[i]);
}


void DVO::init(int w, int h, const Eigen::Matrix3f &K)
{
    // pyramid level size
    int wDown = w;
    int hDown = h;
    int n = wDown*hDown;
    sizePyramid_.push_back(cv::Size(wDown, hDown));

    // gradients
    cv::gpu::GpuMat gradX = cv::gpu::createContinuous(h, w, CV_32FC1);
    gradX_.push_back(gradX);
    cv::gpu::GpuMat gradY = cv::gpu::createContinuous(h, w, CV_32FC1);
    gradY_.push_back(gradY);

    // Jacobian
    float* J;
    hipMalloc(&J,sizeof(float)*n*6);CUDA_CHECK;
    d_J_.push_back(J);
    // residuals
    float* d_residuals;
    hipMalloc(&d_residuals,sizeof(float)*n);CUDA_CHECK;
    d_residuals_.push_back(d_residuals);
    // per-residual weights
    float* weights;
    hipMalloc(&weights,sizeof(float)*n);CUDA_CHECK;
    d_weights_.push_back(weights);

    // camera matrix
    kPyramid_.push_back(K);

    for (int i = 1; i < numPyramidLevels_; ++i)
    {
        // pyramid level size
        wDown = wDown / 2;
        hDown = hDown / 2;
        int n = wDown*hDown;
        sizePyramid_.push_back(cv::Size(wDown, hDown));

        // gradients
        cv::gpu::GpuMat gradXdown = cv::gpu::createContinuous(hDown, wDown, CV_32FC1);
        gradX_.push_back(gradXdown);
        cv::gpu::GpuMat gradYdown = cv::gpu::createContinuous(hDown, wDown, CV_32FC1);
        gradY_.push_back(gradYdown);

        // Jacobian
        float* J;
        hipMalloc(&J,sizeof(float)*n*6);CUDA_CHECK;
        d_J_.push_back(J);
        // residuals
        float* d_residuals;
        hipMalloc(&d_residuals,sizeof(float)*n);CUDA_CHECK;
        d_residuals_.push_back(d_residuals);
        // per-residual weights
        float* weights;
        hipMalloc(&weights,sizeof(float)*n);CUDA_CHECK;
        d_weights_.push_back(weights);

        // downsample camera matrix
        Eigen::Matrix3f kDown = kPyramid_[i-1];
        kDown(0, 2) += 0.5f;
        kDown(1, 2) += 0.5f;
        kDown.topLeftCorner(2, 3) = kDown.topLeftCorner(2, 3) * 0.5f;
        kDown(0, 2) -= 0.5f;
        kDown(1, 2) -= 0.5f;
        kPyramid_.push_back(kDown);
        //std::cout << "Camera matrix (level " << i << "): " << kDown << std::endl;
    }
}


void DVO::convertSE3ToTf(const Vec6f &xi, Eigen::Matrix3f &rot, Eigen::Vector3f &t)
{
    // rotation
    Sophus::SE3f se3 = Sophus::SE3f::exp(xi);
    Eigen::Matrix4f mat = se3.matrix();
    rot = mat.topLeftCorner(3, 3);
    t = mat.topRightCorner(3, 1);
}


void DVO::convertSE3ToTf(const Vec6f &xi, Eigen::Matrix4f &pose)
{
    Sophus::SE3f se3 = Sophus::SE3f::exp(xi);
    pose = se3.matrix();
}


void DVO::convertTfToSE3(const Eigen::Matrix3f &rot, const Eigen::Vector3f &t, Vec6f &xi)
{
    Sophus::SE3f se3(rot, t);
    xi = Sophus::SE3f::log(se3);
}


void DVO::convertTfToSE3(const Eigen::Matrix4f &pose, Vec6f &xi)
{
    Eigen::Matrix3f rot = pose.topLeftCorner(3, 3);
    Eigen::Vector3f t = pose.topRightCorner(3, 1);
    convertTfToSE3(rot, t, xi);
}


__global__ void downsampleGrayKernel(float* out, int w, int h, float* in)
{
	//Compute index
	int x = threadIdx.x + blockDim.x * blockIdx.x;
	int y = threadIdx.y + blockDim.y * blockIdx.y;
	int z = threadIdx.z + blockDim.z * blockIdx.z;
    int wDown = w/2;
    int hDown = h/2;
	//Do bounds check
	if(x<wDown && y<hDown && z<1)
	{
        float sum = 0.0f;
        sum += in[2*y * w + 2*x] * 0.25f;
        sum += in[2*y * w + 2*x+1] * 0.25f;
        sum += in[(2*y+1) * w + 2*x] * 0.25f;
        sum += in[(2*y+1) * w + 2*x+1] * 0.25f;
        out[y*wDown + x] = sum;
	}
}


cv::gpu::GpuMat DVO::downsampleGray(const cv::gpu::GpuMat &gray, int streamIdx)
{
	float * d_in, * d_out;
    int w = gray.cols;
    int h = gray.rows;
    int wDown = w/2;
    int hDown = h/2;
    d_in = (float*)gray.data;

    cv::gpu::GpuMat grayDown = cv::gpu::createContinuous(hDown,wDown,gray.type());
    d_out = (float*)grayDown.data;

    dim3 block = dim3(64,8,1);
    dim3 grid = dim3((w+block.x-1) / block.x,
		(h+block.y - 1) / block.y,
		1);
    downsampleGrayKernel<<<grid,block,0,streams[streamIdx]>>>(d_out, w, h, d_in);
    hipDeviceSynchronize(); CUDA_CHECK;

    return grayDown;
}


__global__ void downsampleDepthKernel(float* out, int w, int h, float* in)
{
	//Compute index
	int x = threadIdx.x + blockDim.x * blockIdx.x;
	int y = threadIdx.y + blockDim.y * blockIdx.y;
	int z = threadIdx.z + blockDim.z * blockIdx.z;
    int wDown = w/2;
    int hDown = h/2;
	//Do bounds check
	if(x<wDown && y<hDown && z<1)
	{
         float d0 = in[2*y * w + 2*x];
         float d1 = in[2*y * w + 2*x+1];
         float d2 = in[(2*y+1) * w + 2*x];
         float d3 = in[(2*y+1) * w + 2*x+1];

         int cnt = 0;
         float sum = 0.0f;
         if (d0 != 0.0f)
         {
             sum += 1.0f / d0;
             ++cnt;
         }
         if (d1 != 0.0f)
         {
             sum += 1.0f / d1;
             ++cnt;
         }
         if (d2 != 0.0f)
         {
             sum += 1.0f / d2;
             ++cnt;
         }
         if (d3 != 0.0f)
         {
             sum += 1.0f / d3;
             ++cnt;
         }

         if (cnt > 0)
         {
             float dInv = sum / float(cnt);
             if (dInv != 0.0f)
             {
                 out[y*wDown + x] = 1.0f / dInv;
                 return;
             }
         }
         //set pixel if we did not enter the inner if-block
         out[y*wDown + x] = 0.0f;
	}
}


cv::gpu::GpuMat DVO::downsampleDepth(const cv::gpu::GpuMat &depth, int streamIdx)
{

    float * d_in, * d_out;
    int w = depth.cols;
    int h = depth.rows;
    int wDown = w/2;
    int hDown = h/2;
    d_in = (float*)depth.data;

    cv::gpu::GpuMat depthDown = cv::gpu::createContinuous(hDown,wDown,depth.type());
    d_out = (float*)depthDown.data;

    dim3 block = dim3(64,8,1);
    dim3 grid = dim3((w+block.x-1) / block.x,
		(h+block.y - 1) / block.y,
		1);
    downsampleDepthKernel<<<grid,block,0,streams[streamIdx]>>>(d_out, w, h, d_in);
    hipDeviceSynchronize(); CUDA_CHECK;

    return depthDown;

}


__global__ void computeGradientKernel(float* outx,float *outy,const float* in, int w, int h)
{
	//Compute index
	int x = threadIdx.x + blockDim.x * blockIdx.x;
	int y = threadIdx.y + blockDim.y * blockIdx.y;
	//Do bounds check
	if(x < w && y < h)
	{
		//if we are out of the specified range but still inside the frame, we need to set
		//the pixel anyway (analog to pre-initialization in the sequential code)
		outx[y*w + x] = 0.0f;
		outy[y*w + x] = 0.0f;

		float v0 = 0.0f;
		float v1 = 0.0f;

		// Along y direction
		if((y-1)>=0 && (y+1)<h){
			v0 = in[(y-1)*w + x];
		    v1 = in[(y+1)*w + x];
		    outy[y*w + x] = 0.5f * (v1 - v0);
		}
		// Along x direction
		if((x-1)>=0 && (x+1)<w) {
			v0 = in[y*w + (x-1)];
		    v1 = in[y*w + (x+1)];
		    outx[y*w + x] = 0.5f * (v1 - v0);
		}
	}

}


void DVO::computeGradient(const cv::gpu::GpuMat &gray, cv::gpu::GpuMat &gradientx, cv::gpu::GpuMat &gradienty)
{

    // compute gradient manually using finite differences
    int w = gray.cols;
    int h = gray.rows;
    const float* d_ptrIn = (const float*)gray.data;
    gradientx.setTo(0);
    gradienty.setTo(0);
    float* d_ptrOutx = (float*)gradientx.data;
    float* d_ptrOuty = (float*)gradienty.data;
    dim3 block = dim3(64,8,1);
    dim3 grid = dim3((w+1+block.x) / block.x,(h+block.y) / block.y,1);
    computeGradientKernel<<<grid,block>>>(d_ptrOutx,d_ptrOuty, d_ptrIn, w, h);
    hipDeviceSynchronize(); CUDA_CHECK;
}


struct is_nonzero : public thrust::unary_function<float,bool>
{
    __host__ __device__
    bool operator()(float x)
    {
        return  x != 0.0f;
    }
};

struct squareop
    : std::unary_function<float, float>
    {
        __host__ __device__ float operator()(float data) {
        	return data*data;
        }
    };


float DVO::calculateError(float* d_residuals, int n)
{
    float error = 0.0f;

    thrust::device_ptr<float> dp_residuals = thrust::device_pointer_cast(d_residuals);

    int numValid = thrust::count_if(dp_residuals,dp_residuals+n, is_nonzero());
    error = thrust::transform_reduce(
    		dp_residuals,
    		dp_residuals+n,
            squareop(),
            0.0f,
            thrust::plus<float>());

    if (numValid > 0)
    	error = error / static_cast<float>(numValid);

    return error;
}

__host__ __device__ float d_interpolate(const float* ptrImgIntensity, float x, float y, int w, int h)
{
    float valCur = nan("");

#if 0
    // direct lookup, no interpolation
    int x0 = static_cast<int>(x + 0.5f);
    int y0 = static_cast<int>(y + 0.5f);
    if (x0 >= 0 && x0 < w && y0 >= 0 && y0 < h)
        valCur = ptrImgIntensity[y0*w + x0];
#else
    //bilinear interpolation
    int x0 = static_cast<int>(x);
    int y0 = static_cast<int>(y);
    int x1 = x0 + 1;
    int y1 = y0 + 1;

    float x1_weight = x - static_cast<float>(x0);
    float y1_weight = y - static_cast<float>(y0);
    float x0_weight = 1.0f - x1_weight;
    float y0_weight = 1.0f - y1_weight;

    if (x0 < 0 || x0 >= w)
        x0_weight = 0.0f;
    if (x1 < 0 || x1 >= w)
        x1_weight = 0.0f;
    if (y0 < 0 || y0 >= h)
        y0_weight = 0.0f;
    if (y1 < 0 || y1 >= h)
        y1_weight = 0.0f;
    float w00 = x0_weight * y0_weight;
    float w10 = x1_weight * y0_weight;
    float w01 = x0_weight * y1_weight;
    float w11 = x1_weight * y1_weight;

    float sumWeights = w00 + w10 + w01 + w11;
    float sum = 0.0f;
    if (w00 > 0.0f)
        sum += ptrImgIntensity[y0*w + x0] * w00;
    if (w01 > 0.0f)
        sum += ptrImgIntensity[y1*w + x0] * w01;
    if (w10 > 0.0f)
        sum += ptrImgIntensity[y0*w + x1] * w10;
    if (w11 > 0.0f)
        sum += ptrImgIntensity[y1*w + x1] * w11;

    if (sumWeights > 0.0f)
        valCur = sum / sumWeights;
#endif

    return valCur;
}



__global__ void g_residualKernel(const float* d_ptrGrayRef,
                            const float* d_ptrDepthRef,
                            const float* d_ptrGrayCur,
                            const float* d_ptrRotation,
                            const float* d_ptrTranslation,
                            float fx, float fy, float cx, float cy, int w,int h,
                            float* d_residuals)
{
    int x = threadIdx.x + blockDim.x*blockIdx.x;
    int y = threadIdx.y + blockDim.y*blockIdx.y;

    // valid thread index
    if(x < w && y < h) {

        size_t idx = x + y*w;
        float residual = 0.0f;

        // backproject 2d pixel
        float dRef = d_ptrDepthRef[idx];

        // continue if valid depth data is available
        if(dRef > 0.0) {
            // to camera coordinates
            float x0 = (static_cast<float>(x) - cx) * 1.0f/fx;
            float y0 = (static_cast<float>(y) - cy) * 1.0f/fy;
            float homo = 1.0f;

            // apply known depth; to 3D coordinates
            x0  *= dRef;
            y0  *= dRef;
            float z0 = homo * dRef;

            // rotate and translate; Eigen uses column-major
            float x1 = d_ptrRotation[0] * x0 + d_ptrRotation[3] * y0 +
                        d_ptrRotation[6] * z0 + d_ptrTranslation[0];
            float y1 = d_ptrRotation[1] * x0 + d_ptrRotation[4] * y0 +
                        d_ptrRotation[7] * z0 + d_ptrTranslation[1];
            float z1 = d_ptrRotation[2] * x0 + d_ptrRotation[5] * y0 +
                        d_ptrRotation[8] * z0 + d_ptrTranslation[2];

            if(z1 > 0.0f) {
                // project onto 2nd frame

                float x2 = (fx * x1 + cx * z1) / z1;
                float y2 = (fy * y1 + cy * z1) / z1;

                float valCur = d_interpolate(d_ptrGrayCur, x2, y2, w, h);
                if (!isnan(valCur))
                {
                    float valRef = d_ptrGrayRef[idx];
                    float valDiff = valRef - valCur;
                    residual = valDiff;
                }

                /*if(x2 >= 0 && x2 < w && y2 >= 0 && y2 < h) {
                    // interpolate
                    float valCur = tex2D(texGrayCur, x2, y2);
                    residual = d_ptrGrayRef[idx] - valCur;
                }*/
            }
        }
        d_residuals[idx] = residual;
    }
}


void DVO::calculateError(const cv::gpu::GpuMat &grayRef, const cv::gpu::GpuMat &depthRef,
                         const cv::gpu::GpuMat &grayCur, const cv::gpu::GpuMat &depthCur,
                         const Eigen::VectorXf &xi, const Eigen::Matrix3f &K,
                         float* d_residuals)
{

    // create residual image
    int w = grayRef.cols;
    int h = grayRef.rows;

    // camera intrinsics
    float fx = K(0, 0);
    float fy = K(1, 1);
    float cx = K(0, 2);
    float cy = K(1, 2);

    // convert SE3 to rotation matrix and translation vector
    Eigen::Matrix3f rotMat;
    Eigen::Vector3f t;
    convertSE3ToTf(xi, rotMat, t);

    float* d_ptrGrayRef = (float*)grayRef.ptr();
    float* d_ptrDepthRef = (float*)depthRef.ptr();
    float* d_ptrGrayCur = (float*)grayCur.ptr();
    float* d_ptrDepthCur = (float*)depthCur.ptr();

    float* d_ptrRotation;
    hipMalloc(&d_ptrRotation, 9*sizeof(float)); CUDA_CHECK;
    hipMemcpy(d_ptrRotation, rotMat.data(), 9*sizeof(float), hipMemcpyHostToDevice);CUDA_CHECK;

    float* d_ptrTranslation;
    hipMalloc(&d_ptrTranslation, 3*sizeof(float));CUDA_CHECK;
    hipMemcpy(d_ptrTranslation, t.data(), 3*sizeof(float), hipMemcpyHostToDevice);CUDA_CHECK;

    dim3 block = dim3(32,8,1);
    dim3 grid = dim3( (w + block.x -1) / block.x, (h+block.y -1) / block.y, 1);
    g_residualKernel <<<grid,block>>> (d_ptrGrayRef, d_ptrDepthRef, d_ptrGrayCur, d_ptrRotation,
                                d_ptrTranslation, fx, fy, cx, cy, w, h, d_residuals);
    hipDeviceSynchronize();


    hipFree(d_ptrRotation);CUDA_CHECK;
    hipFree(d_ptrTranslation);CUDA_CHECK;
    //hipFree(d_residuals);CUDA_CHECK;
    //hipUnbindTexture(texGrayCur);


}


__global__ void computeAndApplyHuberWeightsKernel(float* weights, float* residuals, int n, float k)
{
	//Compute index
	int x = threadIdx.x + blockDim.x * blockIdx.x;
	int y = threadIdx.y + blockDim.y * blockIdx.y;
	int z = threadIdx.z + blockDim.z * blockIdx.z;
	int i = x;
	//Do bounds check
	if(i<n && y < 1 && z < 1)
	{
		//compute robust Huber weights
        float w;
        if (std::abs(residuals[i]) <= k)
            w = 1.0f;
        else
            w = k / std::abs(residuals[i]);
        weights[i] = w;

        //apply weights
        residuals[i] = residuals[i] * weights[i];
	}
}



struct varianceshifteop
    : std::unary_function<float, float>
{
    varianceshifteop(float m)
        : mean(m)
    { /* no-op */ }

    const float mean;

    __device__ float operator()(float data) const
    {
    	return (data-mean)*(data-mean);
    }
};



void DVO::computeAndApplyWeights(float* d_residuals, float* d_weights, int n)
{
#if 0
    // no weighting
    for (int i = 0; i < n; ++i)
        weights[i] = 1.0f;
#if 0
    // squared residuals
    for (int i = 0; i < n; ++i)
        residuals[i] = residuals[i] * residuals[i];
    return;
#endif
#endif

    float mean, stdDev;

    // wrap raw pointer with a device_ptr
    thrust::device_ptr<float> dp_residuals = thrust::device_pointer_cast(d_residuals);

    // sum elements and divide by the number of elements
    mean = thrust::reduce(
        dp_residuals,
        dp_residuals+n,
        0.0f,
        thrust::plus<float>()) / n;

    // shift elements by mean, square, and add them
    float variance = thrust::transform_reduce(
    		dp_residuals,
    		dp_residuals+n,
            varianceshifteop(mean),
            0.0f,
            thrust::plus<float>());

    // standard dev is just a sqrt away
    stdDev = std::sqrt(variance);

    float k = 1.345f * stdDev;

    dim3 block = dim3(512,1,1);
    dim3 grid = dim3((n+block.x-1) / block.x,
		1,
		1);
    computeAndApplyHuberWeightsKernel<<<grid,block>>>(d_weights, d_residuals, n, k);
    hipDeviceSynchronize(); CUDA_CHECK;

}
/*
__global__ void applyWeightsKernel(const float* weights, float* residuals, int n)
{
	//Compute index
	int x = threadIdx.x + blockDim.x * blockIdx.x;
	int y = threadIdx.y + blockDim.y * blockIdx.y;
	int z = threadIdx.z + blockDim.z * blockIdx.z;
	int i = x;
	//Do bounds check
	if(i<n && y < 1 && z < 1)
	{
		residuals[i] = residuals[i] * weights[i];
	}
}


void DVO::applyWeights(const float* d_weights, float* d_residuals, int n)
{

    dim3 block = dim3(512,1,1);
    dim3 grid = dim3((n+block.x-1) / block.x,
		1,
		1);
    applyWeightsKernel<<<grid,block>>>(d_weights, d_residuals, n);
    hipDeviceSynchronize(); CUDA_CHECK;

}
*/

__global__ void computeJtRIntermediateResultKernel(float* out, const float* J, const float* residuals, int m, int j)
{
	//Compute index
	int x = threadIdx.x + blockDim.x * blockIdx.x;
	int y = threadIdx.y + blockDim.y * blockIdx.y;
	int z = threadIdx.z + blockDim.z * blockIdx.z;
	int i = x;
	if(i<m && y < 1 && z < 1)
	{
		out[i] = J[i*6 + j] * residuals[i];
	}
}

void DVO::compute_JtR(float* d_J, const float* d_residuals, Vec6f &b, int validRows)
{

    int n = 6;
    int m = validRows;

    float alpha = 1;
    float beta = 0;

    float *d_y;
    hipMalloc(&d_y,sizeof(float)*6);

    hipblasSgemv(handle,HIPBLAS_OP_N,n,m,&alpha,d_J,n,d_residuals,1,&beta,d_y,1);

    float *res = new float[6];
    hipMemcpy(res,d_y,sizeof(float)*6,hipMemcpyDeviceToHost);

    for(int i = 0; i < 6; i++) {
        b[i] = res[i];
    }

    delete[] res;
    hipFree(d_y);

}

__global__ void JtJKernel(const float* d_J,  const float* d_weights, int validRows, bool useWeights, float *d_res) {

    int i = threadIdx.x + blockDim.x*blockIdx.x;
    int m = blockIdx.y;

    int n = 6;

    int k = floor( ( 2.0f*n+1 - sqrtf( (2.0f*n+1.0f)*(2.0f*n+1.0f) - 8.0f*m ) ) / 2.0f ) ;
    int j = k + (m - n*k + k*(k-1)/2) ;


    if(i < validRows) {

        float valSqr;
    	valSqr = d_J[i*6 + j] * d_J[i*6 + k];
    	if (useWeights)
    	   valSqr *= d_weights[i];

        d_res[i + m*validRows] = valSqr;
    }
}

__global__ void WJKernel(const float* d_J, const float* d_weights, int validRows, float *d_res) {
    int m = threadIdx.x + blockDim.x*blockIdx.x;
    int n = threadIdx.y;

    //extern __shared__ float s_weights[];

    /*if(m < validRows) {
        s_weights[m] = d_weights[m];
    }

    __syncthreads();
*/
    if(m < validRows) {
        d_res[n + m*6] = d_weights[m] * d_J[n + m*6];
    }
}


void DVO::compute_JtJ(const float* d_J, Mat6f &A, const float* d_weights, int validRows, bool useWeights)
{
    /*
    int n = 6;
    int m = validRows;

    dim3 block = dim3(128,1,1);
    // matrix A has 21 unique elements due to symmetry
	dim3 grid = dim3((m+block.x-1) / block.x,21,1);

    float *d_res;
    hipMalloc(&d_res, sizeof(float)*21*m);CUDA_CHECK;
    thrust::device_ptr<float> dp_res = thrust::device_pointer_cast(d_res);


    JtJKernel <<<grid,block>>>(d_J, d_weights, validRows, useWeights, d_res);

    hipStream_t stream0;
    hipStreamCreate ( &stream0) ;

    //float *res = new float[36];

    //hipMemcpy(res,d_res,sizeof(float)*36,hipMemcpyDeviceToHost);

    // column-major

    /*float *res = new float[21*m];
    hipMemcpy(res,d_res,sizeof(float)*21*m,hipMemcpyDeviceToHost);
    ///////
    hipDeviceSynchronize();

    for(int i = 0; i < 21; i++) {
        if(i % 2 == 0) {
            thrust::cuda::par.on(stream0);
        } else {
            thrust::cuda::par.on(0);
        }

        float val = thrust::reduce(
            dp_res + i*m,
            dp_res +i*m +m,
            0.0f,
            thrust::plus<float>());
        /*float val;
        for(int l = 0; l < m; l++) {
            val += res[i*m +l];
        }
        int k = floor( ( 2.0f*n+1 - sqrtf( (2.0f*n+1.0f)*(2.0f*n+1.0f) - 8.0f*i ) ) / 2.0f ) ;
        int j = k + (i - n*k + k*(k-1)/2) ;

        A(j,k) = val;
        A(k,j) = val;
    }
    //delete[] res;
    hipFree(d_res);
*/
/*
    int n = 6;
    int m = validRows;

    float *J = new float[n*m];
    hipMemcpy(J,d_J, sizeof(float)*n*m, hipMemcpyDeviceToHost);

    float *weights = new float[m];
    hipMemcpy(weights,d_weights, sizeof(float)*m, hipMemcpyDeviceToHost);

    // compute A = Jt*J
    for (int k = 0; k < n; ++k)
    {
        for (int j = k; j < n; ++j)
        {
            float val = 0.0f;
            for (int i = 0; i < m; ++i)
            {
                float valSqr = J[i*6 + j] * J[i*6 + k];
                if (useWeights)
                    valSqr *= weights[i];
                val += valSqr;
            }
            A(k, j) = val;
            A(j,k) = val;
        }
    }

    delete[] J;
    delete[] weights;

//    t.end();


//    std::cout << "CPU: " << t.get() << std::endl;
*/
    int n = 6;
    int m = validRows;

    float *d_res;
    hipMalloc(&d_res,sizeof(float)*36);

    float alpha = 1;
    float beta = 0;


    if(useWeights) {
        float *d_WJ;
        hipMalloc(&d_WJ,sizeof(float)*n*m);

        dim3 block = dim3(32,6,1);
        dim3 grid = dim3( (m + block.x -1) / block.x, 1, 1);

        WJKernel <<<grid, block>>> (d_J, d_weights, validRows,d_WJ);
        //hipDeviceSynchronize(); CUDA_CHECK;

        //hipblasSdgmm(handle, HIPBLAS_SIDE_RIGHT, n,m, d_J, n, d_weights, 0, d_WJ, n);
        hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, n, n, m, &alpha, d_WJ, n, d_J, n, &beta, d_res, n);
        hipFree(d_WJ);

    } else {
        hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, n, n, m, &alpha, d_J, n, d_J, n, &beta, d_res, n);
    }

    // column-major
    float *res = new float[36];
    hipMemcpy(res,d_res,sizeof(float)*36,hipMemcpyDeviceToHost);

    // column-major
    for(int k = 0; k < n; k++) {
        for(int j = 0; j < n; j++) {
            A(j,k) = res[k + 6*j];
        }
    }

    delete[] res;
    hipFree(d_res);


}

__device__ void rotateAndTranslate(float *rot,float *t, float *v, float *res)
{
	for(int i = 0;i<3;i++)
	{
		float sum = 0.f;
		for(int j = 0;j<3;j++)
		{
			sum += rot[i+3*j]*v[j];
		}
		res[i] = sum + t[i];
	}

}


__global__ void computeAnalyticalGradient(float *d_K,float* d_ptrDepthRef,float * d_rotMat, float* d_t,
		float *d_gradx,float *d_grady,int w, int h,float *d_J)
{

	int x = threadIdx.x + blockDim.x*blockIdx.x;
	int y = threadIdx.y + blockDim.y*blockIdx.y;


	if(x<w && y<h)
	{
		size_t idx = x + (size_t)w*y;

		float fx = d_K[0];
		float fy = d_K[4];
		float cx = d_K[6];
		float cy = d_K[7];
		float fxInv = 1.0f / fx;
    	float fyInv = 1.0f / fy;

		bool innerIfExecuted=false;

        // project 2d point back into 3d using its depth
        float dRef = d_ptrDepthRef[idx];
        if (dRef > 0.0f)
        {
            float x0 = (static_cast<float>(x) - cx) * fxInv;
            float y0 = (static_cast<float>(y) - cy) * fyInv;
            float scale = 1.0f;
            //scale = std::sqrt(x0*x0 + y0*y0 + 1.0);
            dRef = dRef * scale;
            x0 = x0 * dRef;
            y0 = y0 * dRef;

            // transform reference 3d point into current frame
            // reference 3d point
            // Eigen::Vector3f pt3Ref(x0, y0, dRef);
            float pt3Ref[3] = {x0,y0,dRef};
            float pt3[3];

            rotateAndTranslate(d_rotMat,d_t,pt3Ref,pt3);

            if (pt3[2] > 0.0f)
            {
/*
                // project 3d point to 2d
                float pt2CurH[3];
                multiply(d_K,pt3,pt2CurH);
            	//Eigen::Vector3f pt2CurH = K * pt3;

                float ptZinv = 1.0f / pt2CurH[2];
                float px = pt2CurH[0] * ptZinv;
                float py = pt2CurH[1] * ptZinv;
                */

                float px = (fx * pt3[0] + cx * pt3[2]) / pt3[2];
                float py = (fy * pt3[1] + cy * pt3[2]) / pt3[2];

                // compute interpolated image gradient
                float dX = d_interpolate(d_gradx, px, py, w, h);
                float dY = d_interpolate(d_grady, px, py, w, h);

                if (!isnan(dX) && !isnan(dY))
                {
                	innerIfExecuted = true;
                    dX = fx * dX;
                    dY = fy * dY;
                    float pt3Zinv = 1.0f / pt3[2];

                    // shorter computation
                   		d_J[idx*6 + 0] = -1.0f*dX * pt3Zinv;
		                d_J[idx*6 + 1] = -1.0f*dY * pt3Zinv;
		                d_J[idx*6 + 2] = (dX * pt3[0] + dY * pt3[1]) * pt3Zinv * pt3Zinv;
		                d_J[idx*6 + 3] = (dX * pt3[0] * pt3[1]) * pt3Zinv * pt3Zinv + dY * (1 + (pt3[1] * pt3Zinv) * (pt3[1] * pt3Zinv));
		                d_J[idx*6 + 4] = - dX * (1.0 + (pt3[0] * pt3Zinv) * (pt3[0] * pt3Zinv)) - (dY * pt3[0] * pt3[1]) * pt3Zinv * pt3Zinv;
		                d_J[idx*6 + 5] = -1.0f*(- dX * pt3[1] + dY * pt3[0]) * pt3Zinv;
                }
            }
        }


        if(!innerIfExecuted) {
        	for (int j=0;j<6;j++) {

        			d_J[idx*6 + j] =  0.0f;

        	}
        }

    }
}


void DVO::deriveAnalytic(const cv::gpu::GpuMat &grayRef, const cv::gpu::GpuMat &depthRef,
                   const cv::gpu::GpuMat &grayCur, const cv::gpu::GpuMat &depthCur,
                   const cv::gpu::GpuMat &gradX, const cv::gpu::GpuMat &gradY,
                   const Eigen::VectorXf &xi, const Eigen::Matrix3f &K,
                   float* d_residuals, float* d_J)
{
    // reference input images
    int w = grayRef.cols;
    int h = grayRef.rows;
    int n = w*h;

    // camera intrinsics
    float fx = K(0, 0);
    float fy = K(1, 1);
    float cx = K(0, 2);
    float cy = K(1, 2);

    // convert SE3 to rotation matrix and translation vector
    Eigen::Matrix3f rotMat;
    Eigen::Vector3f t;
    convertSE3ToTf(xi, rotMat, t);

    float* d_ptrGrayRef = (float*)grayRef.ptr();
    float* d_ptrDepthRef = (float*)depthRef.ptr();
    float* d_ptrGrayCur = (float*)grayCur.ptr();
    float* d_ptrDepthCur = (float*)depthCur.ptr();

    // Allocating device memory
    float *d_gradx,*d_grady,*d_t,*d_K,*d_rotMat;

    d_gradx = (float*) gradX.data;
    d_grady = (float*) gradY.data;

    hipMalloc(&d_rotMat,9*sizeof(float));CUDA_CHECK;
    hipMalloc(&d_K,9*sizeof(float));CUDA_CHECK;
    hipMalloc(&d_t,3*sizeof(float));CUDA_CHECK;

    hipMemcpy(d_rotMat,rotMat.data(),9*sizeof(float),hipMemcpyHostToDevice);CUDA_CHECK;
    hipMemcpy(d_K,K.data(),9*sizeof(float),hipMemcpyHostToDevice);CUDA_CHECK;
    hipMemcpy(d_t,t.data(),3*sizeof(float),hipMemcpyHostToDevice);CUDA_CHECK;


    dim3 block = dim3(32,8,1);
    dim3 grid = dim3( (w + block.x -1) / block.x, (h+block.y -1) / block.y, 1);

    hipStream_t stream0, stream1;
    hipStreamCreate ( &stream0) ;
    hipStreamCreate ( &stream1) ;

    g_residualKernel <<<grid,block,0,stream0>>> (d_ptrGrayRef, d_ptrDepthRef, d_ptrGrayCur, d_rotMat,
                                d_t, fx, fy, cx, cy, w, h, d_residuals);
    computeAnalyticalGradient<<<grid,block,0,stream1>>>(d_K,d_ptrDepthRef,d_rotMat,d_t,d_gradx,d_grady,w,h,d_J);
    hipDeviceSynchronize();

}


cv::gpu::GpuMat DVO::convertToContGpuMat(const cv::Mat &m) {
    cv::gpu::GpuMat gpuM = cv::gpu::createContinuous(m.rows, m.cols, m.type());
    gpuM.upload(m);
    return gpuM;
}

void DVO::buildPyramid(const cv::Mat &depth, const cv::Mat &gray, std::vector<cv::gpu::GpuMat> &depthPyramid, std::vector<cv::gpu::GpuMat> &grayPyramid)
{
    grayPyramid.push_back(convertToContGpuMat(gray));
    depthPyramid.push_back(convertToContGpuMat(depth));

    for (int i = 1; i < numPyramidLevels_; ++i)
    {
        // downsample grayscale image
        cv::gpu::GpuMat grayDown = downsampleGray(grayPyramid[i-1], 0);
        // downsample depth image
        cv::gpu::GpuMat depthDown = downsampleDepth(depthPyramid[i-1], 1);
        hipDeviceSynchronize(); CUDA_CHECK;

        grayPyramid.push_back(grayDown);
        depthPyramid.push_back(depthDown);
    }
}


void DVO::align(const cv::Mat &depthRef, const cv::Mat &grayRef, const cv::Mat &depthCur, const cv::Mat &grayCur, Eigen::Matrix4f &pose)
{
    // downsampling

    std::vector<cv::gpu::GpuMat> grayRefGPUPyramid;
    std::vector<cv::gpu::GpuMat> depthRefGPUPyramid;

    buildPyramid(depthRef, grayRef, depthRefGPUPyramid, grayRefGPUPyramid);

    std::vector<cv::gpu::GpuMat> grayCurGPUPyramid;
    std::vector<cv::gpu::GpuMat> depthCurGPUPyramid;

    buildPyramid(depthCur, grayCur, depthCurGPUPyramid, grayCurGPUPyramid);

    align(depthRefGPUPyramid, grayRefGPUPyramid, depthCurGPUPyramid, grayCurGPUPyramid, pose);
}


void DVO::align(const std::vector<cv::gpu::GpuMat> &depthRefGPUPyramid, const std::vector<cv::gpu::GpuMat> &grayRefGPUPyramid,
                const std::vector<cv::gpu::GpuMat> &depthCurGPUPyramid, const std::vector<cv::gpu::GpuMat> &grayCurGPUPyramid,
                Eigen::Matrix4f &pose)
{

    Vec6f xi;
    convertTfToSE3(pose, xi);

    Vec6f lastXi = Vec6f::Zero();

    int maxLevel = numPyramidLevels_-1;
    int minLevel = 1;
    float initGradDescStepSize = 1e-3f;
    float gradDescStepSize = initGradDescStepSize;

    Mat6f A;
    Mat6f diagMatA = Mat6f::Identity();
    Vec6f delta;

    for (int lvl = maxLevel; lvl >= minLevel; --lvl)
    {
        float lambda = 0.1f;

        int w = sizePyramid_[lvl].width;
        int h = sizePyramid_[lvl].height;
        int n = w*h;

        cv::gpu::GpuMat grayRef = grayRefGPUPyramid[lvl];
        cv::gpu::GpuMat depthRef = depthRefGPUPyramid[lvl];
        cv::gpu::GpuMat grayCur = grayCurGPUPyramid[lvl];
        cv::gpu::GpuMat depthCur = depthCurGPUPyramid[lvl];
        Eigen::Matrix3f kLevel = kPyramid_[lvl];
        //std::cout << "level " << level << " (size " << depthRef.cols << "x" << depthRef.rows << ")" << std::endl;

        // compute gradient images
        computeGradient(grayCur, gradX_[lvl],gradY_[lvl]);
        //computeGradient(grayCur, gradY_[lvl], 1);

        float errorLast = std::numeric_limits<float>::max();
        for (int itr = 0; itr < numIterations_; ++itr)
        {
            // compute residuals and Jacobian
#if 0
            deriveNumeric(grayRef, depthRef, grayCur, depthCur, xi, kLevel, residuals_[lvl], J_[lvl]);
#else
            deriveAnalytic(grayRef, depthRef, grayCur, depthCur, gradX_[lvl], gradY_[lvl], xi, kLevel, d_residuals_[lvl], d_J_[lvl]);
#endif

#if 0
            // compute and show error image
            cv::Mat errorImage;
            calculateErrorImage(residuals_[lvl], grayRef.cols, grayRef.rows, errorImage);
            std::stringstream ss;
            ss << "residuals_" << lvl << "_";
            ss << std::setw(2) << std::setfill('0') << itr << ".png";
            cv::imwrite(ss.str(), errorImage);
            cv::imshow("error", errorImage);
            cv::waitKey(0);
#endif

            // calculate error
            float error = calculateError(d_residuals_[lvl], n);
            if (useWeights_)
            {
                // compute and apply robust weights
                computeAndApplyWeights(d_residuals_[lvl], d_weights_[lvl], n);
            }

            // compute update
            Vec6f b;
            compute_JtR(d_J_[lvl], d_residuals_[lvl], b, n);

            if (algo_ == GradientDescent)
            {
                // Gradient Descent
                delta = -gradDescStepSize * b * (1.0f / b.norm());
            }
            else if (algo_ == GaussNewton)
            {
                // Gauss-Newton algorithm
                compute_JtJ(d_J_[lvl], A, d_weights_[lvl], n, useWeights_);
                // solve using Cholesky LDLT decomposition
                delta = -(A.ldlt().solve(b));
            }
            else if (algo_ == LevenbergMarquardt)
            {
                // Levenberg-Marquardt algorithm
                compute_JtJ(d_J_[lvl], A, d_weights_[lvl], n, useWeights_);
                diagMatA.diagonal() = lambda * A.diagonal();
                delta = -((A + diagMatA).ldlt().solve(b));
            }

            // apply update: left-multiplicative increment on SE3
            lastXi = xi;
            xi = Sophus::SE3f::log(Sophus::SE3f::exp(delta) * Sophus::SE3f::exp(xi));
#if 0
            std::cout << "delta = " << delta.transpose() << " size = " << delta.rows() << " x " << delta.cols() << std::endl;
            std::cout << "xi = " << xi.transpose() << std::endl;
#endif

            // compute error again
            error = calculateError(d_residuals_[lvl], n);

            if (algo_ == LevenbergMarquardt)
            {
                if (error >= errorLast)
                {
                    lambda = lambda * 5.0f;
                    xi = lastXi;

                    if (lambda > 5.0f)
                        break;
                }
                else
                {
                    lambda = lambda / 1.5f;
                }
            }
            else if (algo_ == GaussNewton)
            {
                // break if no improvement (0.99 or 0.995)
                if (error / errorLast > 0.995f)
                    break;
            }
            else if (algo_ == GradientDescent)
            {
                if (error >= errorLast)
                {
                    gradDescStepSize = gradDescStepSize * 0.5f;
                    if (gradDescStepSize <= initGradDescStepSize * 0.01f)
                        gradDescStepSize = initGradDescStepSize * 0.01f;
                    xi = lastXi;
                }
                else
                {
                    gradDescStepSize = gradDescStepSize * 2.0f;
                    if (gradDescStepSize >= initGradDescStepSize * 100.0f)
                        gradDescStepSize = initGradDescStepSize * 100.0f;

                    // break if no improvement (0.99 or 0.995)
                    if (error / errorLast > 0.995f)
                        break;
                }
            }

            errorLast = error;
        }
    }

    // store to output pose
    convertSE3ToTf(xi, pose);
}
