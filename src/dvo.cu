#include "hip/hip_runtime.h"
// Copyright 2016 Robert Maier, Technical University Munich
#include "dvo.hpp"
#include "helper.h"

#include <iostream>
#include <sstream>
#include <string>
#include <iomanip>
#include <stdio.h>
#include <ctime>

#include "helper.h"

#include <Eigen/Cholesky>
#include <sophus/se3.hpp>

#include <opencv2/imgproc/imgproc.hpp>
#include <opencv2/highgui/highgui.hpp>

#include <hip/hip_runtime.h>
#include <thrust/count.h>
#include <hipblas.h>

#include <math.h>
#include <thrust/execution_policy.h>


#define JTR_USE_INNER_PRODUCT

DVO::DVO() :
    numPyramidLevels_(5),
    useWeights_(true),
    numIterations_(500),
    algo_(GaussNewton)
{
}


DVO::~DVO()
{
    for (int i = 0; i < numPyramidLevels_; ++i)
    {
        hipFree(d_J_[i]); CUDA_CHECK;
        hipFree(d_residuals_[i]);CUDA_CHECK;
        hipFree(d_weights_[i]);CUDA_CHECK;
    }
}


void DVO::init(int w, int h, const Eigen::Matrix3f &K)
{
    // pyramid level size
    int wDown = w;
    int hDown = h;
    int n = wDown*hDown;
    sizePyramid_.push_back(cv::Size(wDown, hDown));

    // gradients
    cv::gpu::GpuMat gradX = cv::gpu::createContinuous(h, w, CV_32FC1);
    gradX_.push_back(gradX);
    cv::gpu::GpuMat gradY = cv::gpu::createContinuous(h, w, CV_32FC1);
    gradY_.push_back(gradY);

    // Jacobian
    float* J;
    hipMalloc(&J,sizeof(float)*n*6);CUDA_CHECK;
    d_J_.push_back(J);
    // residuals
    float* d_residuals;
    hipMalloc(&d_residuals,sizeof(float)*n);CUDA_CHECK;
    d_residuals_.push_back(d_residuals);
    // per-residual weights
    float* weights;
    hipMalloc(&weights,sizeof(float)*n);CUDA_CHECK;
    d_weights_.push_back(weights);

    // camera matrix
    kPyramid_.push_back(K);

    for (int i = 1; i < numPyramidLevels_; ++i)
    {
        // pyramid level size
        wDown = wDown / 2;
        hDown = hDown / 2;
        int n = wDown*hDown;
        sizePyramid_.push_back(cv::Size(wDown, hDown));

        // gradients
        cv::gpu::GpuMat gradXdown = cv::gpu::createContinuous(hDown, wDown, CV_32FC1);
        gradX_.push_back(gradXdown);
        cv::gpu::GpuMat gradYdown = cv::gpu::createContinuous(hDown, wDown, CV_32FC1);
        gradY_.push_back(gradYdown);

        // Jacobian
        float* J;
        hipMalloc(&J,sizeof(float)*n*6);CUDA_CHECK;
        d_J_.push_back(J);
        // residuals
        float* d_residuals;
        hipMalloc(&d_residuals,sizeof(float)*n);CUDA_CHECK;
        d_residuals_.push_back(d_residuals);
        // per-residual weights
        float* weights;
        hipMalloc(&weights,sizeof(float)*n);CUDA_CHECK;
        d_weights_.push_back(weights);

        // downsample camera matrix
        Eigen::Matrix3f kDown = kPyramid_[i-1];
        kDown(0, 2) += 0.5f;
        kDown(1, 2) += 0.5f;
        kDown.topLeftCorner(2, 3) = kDown.topLeftCorner(2, 3) * 0.5f;
        kDown(0, 2) -= 0.5f;
        kDown(1, 2) -= 0.5f;
        kPyramid_.push_back(kDown);
        //std::cout << "Camera matrix (level " << i << "): " << kDown << std::endl;
    }
}


void DVO::convertSE3ToTf(const Vec6f &xi, Eigen::Matrix3f &rot, Eigen::Vector3f &t)
{
    // rotation
    Sophus::SE3f se3 = Sophus::SE3f::exp(xi);
    Eigen::Matrix4f mat = se3.matrix();
    rot = mat.topLeftCorner(3, 3);
    t = mat.topRightCorner(3, 1);
}


void DVO::convertSE3ToTf(const Vec6f &xi, Eigen::Matrix4f &pose)
{
    Sophus::SE3f se3 = Sophus::SE3f::exp(xi);
    pose = se3.matrix();
}


void DVO::convertTfToSE3(const Eigen::Matrix3f &rot, const Eigen::Vector3f &t, Vec6f &xi)
{
    Sophus::SE3f se3(rot, t);
    xi = Sophus::SE3f::log(se3);
}


void DVO::convertTfToSE3(const Eigen::Matrix4f &pose, Vec6f &xi)
{
    Eigen::Matrix3f rot = pose.topLeftCorner(3, 3);
    Eigen::Vector3f t = pose.topRightCorner(3, 1);
    convertTfToSE3(rot, t, xi);
}


__global__ void downsampleGrayKernel(float* out, int w, int h, float* in)
{
	//Compute index
	int x = threadIdx.x + blockDim.x * blockIdx.x;
	int y = threadIdx.y + blockDim.y * blockIdx.y;
	int z = threadIdx.z + blockDim.z * blockIdx.z;
    int wDown = w/2;
    int hDown = h/2;
	//Do bounds check
	if(x<wDown && y<hDown && z<1)
	{
        float sum = 0.0f;
        sum += in[2*y * w + 2*x] * 0.25f;
        sum += in[2*y * w + 2*x+1] * 0.25f;
        sum += in[(2*y+1) * w + 2*x] * 0.25f;
        sum += in[(2*y+1) * w + 2*x+1] * 0.25f;
        out[y*wDown + x] = sum;
	}
}


cv::gpu::GpuMat DVO::downsampleGray(const cv::gpu::GpuMat &gray)
{
	float * d_in, * d_out;
    int w = gray.cols;
    int h = gray.rows;
    int wDown = w/2;
    int hDown = h/2;
    d_in = (float*)gray.data;

    cv::gpu::GpuMat grayDown = cv::gpu::createContinuous(hDown,wDown,gray.type());
    d_out = (float*)grayDown.data;

    dim3 block = dim3(64,8,1);
    dim3 grid = dim3((w+block.x-1) / block.x,
		(h+block.y - 1) / block.y,
		1);
    downsampleGrayKernel<<<grid,block>>>(d_out, w, h, d_in);
    hipDeviceSynchronize(); CUDA_CHECK;

    return grayDown;
}


__global__ void downsampleDepthKernel(float* out, int w, int h, float* in)
{
	//Compute index
	int x = threadIdx.x + blockDim.x * blockIdx.x;
	int y = threadIdx.y + blockDim.y * blockIdx.y;
	int z = threadIdx.z + blockDim.z * blockIdx.z;
    int wDown = w/2;
    int hDown = h/2;
	//Do bounds check
	if(x<wDown && y<hDown && z<1)
	{
         float d0 = in[2*y * w + 2*x];
         float d1 = in[2*y * w + 2*x+1];
         float d2 = in[(2*y+1) * w + 2*x];
         float d3 = in[(2*y+1) * w + 2*x+1];

         int cnt = 0;
         float sum = 0.0f;
         if (d0 != 0.0f)
         {
             sum += 1.0f / d0;
             ++cnt;
         }
         if (d1 != 0.0f)
         {
             sum += 1.0f / d1;
             ++cnt;
         }
         if (d2 != 0.0f)
         {
             sum += 1.0f / d2;
             ++cnt;
         }
         if (d3 != 0.0f)
         {
             sum += 1.0f / d3;
             ++cnt;
         }

         if (cnt > 0)
         {
             float dInv = sum / float(cnt);
             if (dInv != 0.0f)
             {
                 out[y*wDown + x] = 1.0f / dInv;
                 return;
             }
         }
         //set pixel if we did not enter the inner if-block
         out[y*wDown + x] = 0.0f;
	}
}


cv::gpu::GpuMat DVO::downsampleDepth(const cv::gpu::GpuMat &depth)
{

    float * d_in, * d_out;
    int w = depth.cols;
    int h = depth.rows;
    int wDown = w/2;
    int hDown = h/2;
    d_in = (float*)depth.data;

    cv::gpu::GpuMat depthDown = cv::gpu::createContinuous(hDown,wDown,depth.type());
    d_out = (float*)depthDown.data;

    dim3 block = dim3(64,8,1);
    dim3 grid = dim3((w+block.x-1) / block.x,
		(h+block.y - 1) / block.y,
		1);
    downsampleDepthKernel<<<grid,block>>>(d_out, w, h, d_in);
    hipDeviceSynchronize(); CUDA_CHECK;

    return depthDown;

}


__global__ void computeGradientKernel(float* out,const float* in, int w, int h,
		int xStart, int yStart, int xEnd, int yEnd, int direction)
{
	//Compute index
	int x = threadIdx.x + blockDim.x * blockIdx.x;
	int y = threadIdx.y + blockDim.y * blockIdx.y;
	int z = threadIdx.z + blockDim.z * blockIdx.z;
	//Do bounds check
	if(xStart <= x && x < xEnd && yStart <= y && y < yEnd && z < 1)
	{
        float v0;
        float v1;
        if (direction == 1)
        {
            // y-direction
            v0 = in[(y-1)*w + x];
            v1 = in[(y+1)*w + x];
        }
        else
        {
            // x-direction
            v0 = in[y*w + (x-1)];
            v1 = in[y*w + (x+1)];
        }
        out[y*w + x] = 0.5f * (v1 - v0);

	}
	//if we are out of the specified range but still inside the frame, we need to set
	//the pixel anyway (analog to pre-initialization in the sequential code)
	else if(x < w && y < h)
	{
		out[y*w + x] = 0.0f;
	}
}


void DVO::computeGradient(const cv::gpu::GpuMat &gray, cv::gpu::GpuMat &gradient, int direction)
{
    int dirX = 1;
    int dirY = 0;
    if (direction == 1)
    {
        dirX = 0;
        dirY = 1;
    }

    // compute gradient manually using finite differences
    int w = gray.cols;
    int h = gray.rows;
    const float* d_ptrIn = (const float*)gray.data;
    gradient.setTo(0);
    float* d_ptrOut = (float*)gradient.data;

    int yStart = dirY;
    int yEnd = h - dirY;
    int xStart = dirX;
    int xEnd = w - dirX;

    dim3 block = dim3(64,8,1);
    dim3 grid = dim3((w+1+block.x-1) / block.x,
		(h+1+block.y - 1) / block.y,
		1);
    computeGradientKernel<<<grid,block>>>(d_ptrOut, d_ptrIn, w, h, xStart, yStart, xEnd, yEnd, direction);
    hipDeviceSynchronize(); CUDA_CHECK;
}


struct is_nonzero : public thrust::unary_function<float,bool>
{
    __host__ __device__
    bool operator()(float x)
    {
        return  x != 0.0f;
    }
};

struct squareop
    : std::unary_function<float, float>
    {
        __host__ __device__ float operator()(float data) {
        	return data*data;
        }
    };


float DVO::calculateError(float* d_residuals, int n)
{
    float error = 0.0f;

    thrust::device_ptr<float> dp_residuals = thrust::device_pointer_cast(d_residuals);

    int numValid = thrust::count_if(dp_residuals,dp_residuals+n, is_nonzero());
    error = thrust::transform_reduce(
    		dp_residuals,
    		dp_residuals+n,
            squareop(),
            0.0f,
            thrust::plus<float>());

    if (numValid > 0)
    	error = error / static_cast<float>(numValid);

    return error;
}


void DVO::calculateErrorImage(const float* residuals, int w, int h, cv::Mat &errorImage)
{
    cv::Mat imgResiduals = cv::Mat::zeros(h, w, CV_32FC1);
    float* ptrResiduals = (float*)imgResiduals.data;

    // fill residuals image
    for (size_t y = 0; y < h; ++y)
    {
        for (size_t x = 0; x < w; ++x)
        {
            size_t off = y*w + x;
            if (residuals[off] != 0.0f)
                ptrResiduals[off] = residuals[off];
        }
    }

    imgResiduals.convertTo(errorImage, CV_8SC1, 127.0);
}



__host__ __device__ float d_interpolate(const float* ptrImgIntensity, float x, float y, int w, int h)
{
    float valCur = nan("");

#if 0
    // direct lookup, no interpolation
    int x0 = static_cast<int>(x + 0.5f);
    int y0 = static_cast<int>(y + 0.5f);
    if (x0 >= 0 && x0 < w && y0 >= 0 && y0 < h)
        valCur = ptrImgIntensity[y0*w + x0];
#else
    //bilinear interpolation
    int x0 = static_cast<int>(x);
    int y0 = static_cast<int>(y);
    int x1 = x0 + 1;
    int y1 = y0 + 1;

    float x1_weight = x - static_cast<float>(x0);
    float y1_weight = y - static_cast<float>(y0);
    float x0_weight = 1.0f - x1_weight;
    float y0_weight = 1.0f - y1_weight;

    if (x0 < 0 || x0 >= w)
        x0_weight = 0.0f;
    if (x1 < 0 || x1 >= w)
        x1_weight = 0.0f;
    if (y0 < 0 || y0 >= h)
        y0_weight = 0.0f;
    if (y1 < 0 || y1 >= h)
        y1_weight = 0.0f;
    float w00 = x0_weight * y0_weight;
    float w10 = x1_weight * y0_weight;
    float w01 = x0_weight * y1_weight;
    float w11 = x1_weight * y1_weight;

    float sumWeights = w00 + w10 + w01 + w11;
    float sum = 0.0f;
    if (w00 > 0.0f)
        sum += ptrImgIntensity[y0*w + x0] * w00;
    if (w01 > 0.0f)
        sum += ptrImgIntensity[y1*w + x0] * w01;
    if (w10 > 0.0f)
        sum += ptrImgIntensity[y0*w + x1] * w10;
    if (w11 > 0.0f)
        sum += ptrImgIntensity[y1*w + x1] * w11;

    if (sumWeights > 0.0f)
        valCur = sum / sumWeights;
#endif

    return valCur;
}


texture<float,2,hipReadModeElementType> texGrayCur;
__global__ void g_residualKernel(const float* d_ptrGrayRef,
                            const float* d_ptrDepthRef,
                            const float* d_ptrGrayCur,
                            const float* d_ptrRotation,
                            const float* d_ptrTranslation,
                            float fx, float fy, float cx, float cy, int w,int h,
                            float* d_residuals)
{
    int x = threadIdx.x + blockDim.x*blockIdx.x;
    int y = threadIdx.y + blockDim.y*blockIdx.y;

    // valid thread index
    if(x < w && y < h) {

        size_t idx = x + y*w;
        float residual = 0.0f;

        // backproject 2d pixel
        float dRef = d_ptrDepthRef[idx];

        // continue if valid depth data is available
        if(dRef > 0.0) {
            // to camera coordinates
            float x0 = (static_cast<float>(x) - cx) * 1.0f/fx;
            float y0 = (static_cast<float>(y) - cy) * 1.0f/fy;
            float homo = 1.0f;

            // apply known depth; to 3D coordinates
            x0  *= dRef;
            y0  *= dRef;
            float z0 = homo * dRef;

            // rotate and translate; Eigen uses column-major
            float x1 = d_ptrRotation[0] * x0 + d_ptrRotation[3] * y0 +
                        d_ptrRotation[6] * z0 + d_ptrTranslation[0];
            float y1 = d_ptrRotation[1] * x0 + d_ptrRotation[4] * y0 +
                        d_ptrRotation[7] * z0 + d_ptrTranslation[1];
            float z1 = d_ptrRotation[2] * x0 + d_ptrRotation[5] * y0 +
                        d_ptrRotation[8] * z0 + d_ptrTranslation[2];

            if(z1 > 0.0f) {
                // project onto 2nd frame

                float x2 = (fx * x1 + cx * z1) / z1;
                float y2 = (fy * y1 + cy * z1) / z1;

                float valCur = d_interpolate(d_ptrGrayCur, x2, y2, w, h);
                if (!isnan(valCur))
                {
                    float valRef = d_ptrGrayRef[idx];
                    float valDiff = valRef - valCur;
                    residual = valDiff;
                }

                /*if(x2 >= 0 && x2 < w && y2 >= 0 && y2 < h) {
                    // interpolate
                    float valCur = tex2D(texGrayCur, x2, y2);
                    residual = d_ptrGrayRef[idx] - valCur;
                }*/
            }
        }
        d_residuals[idx] = residual;
    }
}


void DVO::calculateError(const cv::Mat &grayRef, const cv::Mat &depthRef,
                         const cv::Mat &grayCur, const cv::Mat &depthCur,
                         const Eigen::VectorXf &xi, const Eigen::Matrix3f &K,
                         float* residuals)
{
    // create residual image
    int w = grayRef.cols;
    int h = grayRef.rows;

    // camera intrinsics
    float fx = K(0, 0);
    float fy = K(1, 1);
    float cx = K(0, 2);
    float cy = K(1, 2);
    float fxInv = 1.0f / fx;
    float fyInv = 1.0f / fy;

    // convert SE3 to rotation matrix and translation vector
    Eigen::Matrix3f rotMat;
    Eigen::Vector3f t;
    convertSE3ToTf(xi, rotMat, t);

    const float* ptrGrayRef = (const float*)grayRef.data;
    const float* ptrDepthRef = (const float*)depthRef.data;
    const float* ptrGrayCur = (const float*)grayCur.data;
    const float* ptrDepthCur = (const float*)depthCur.data;

    for (size_t y = 0; y < h; ++y)
    {
        for (size_t x = 0; x < w; ++x)
        {
            size_t off = y*w + x;
            float residual = 0.0f;

            // project 2d point back into 3d using its depth
            float dRef = ptrDepthRef[y*w + x];
            if (dRef > 0.0)
            {
                float x0 = (static_cast<float>(x) - cx) * fxInv;
                float y0 = (static_cast<float>(y) - cy) * fyInv;
                float scale = 1.0f;
                //scale = std::sqrt(x0*x0 + y0*y0 + 1.0f);
                dRef = dRef * scale;
                x0 = x0 * dRef;
                y0 = y0 * dRef;

                // transform reference 3d point into current frame
                // reference 3d point
                Eigen::Vector3f pt3Ref(x0, y0, dRef);
                Eigen::Vector3f pt3Cur = rotMat * pt3Ref + t;
                if (pt3Cur[2] > 0.0f)
                {
                    // project 3d point to 2d
                    Eigen::Vector3f pt2CurH = K * pt3Cur;
                    float ptZinv = 1.0f / pt2CurH[2];
                    float px = pt2CurH[0] * ptZinv;
                    float py = pt2CurH[1] * ptZinv;

                    // interpolate residual
                    float valCur = d_interpolate(ptrGrayCur, px, py, w, h);
                    if (!std::isnan(valCur))
                    {
                        float valRef = ptrGrayRef[off];
                        float valDiff = valRef - valCur;
                        residual = valDiff;
                    }
                }
            }
            residuals[off] = residual;
        }
    }
}

void DVO::calculateError(const cv::gpu::GpuMat &grayRef, const cv::gpu::GpuMat &depthRef,
                         const cv::gpu::GpuMat &grayCur, const cv::gpu::GpuMat &depthCur,
                         const Eigen::VectorXf &xi, const Eigen::Matrix3f &K,
                         float* d_residuals)
{

    // create residual image
    int w = grayRef.cols;
    int h = grayRef.rows;

    // camera intrinsics
    float fx = K(0, 0);
    float fy = K(1, 1);
    float cx = K(0, 2);
    float cy = K(1, 2);

    // convert SE3 to rotation matrix and translation vector
    Eigen::Matrix3f rotMat;
    Eigen::Vector3f t;
    convertSE3ToTf(xi, rotMat, t);

    float* d_ptrGrayRef = (float*)grayRef.ptr();
    float* d_ptrDepthRef = (float*)depthRef.ptr();
    float* d_ptrGrayCur = (float*)grayCur.ptr();
    float* d_ptrDepthCur = (float*)depthCur.ptr();

    float* d_ptrRotation;
    hipMalloc(&d_ptrRotation, 9*sizeof(float)); CUDA_CHECK;
    hipMemcpy(d_ptrRotation, rotMat.data(), 9*sizeof(float), hipMemcpyHostToDevice);CUDA_CHECK;

    float* d_ptrTranslation;
    hipMalloc(&d_ptrTranslation, 3*sizeof(float));CUDA_CHECK;
    hipMemcpy(d_ptrTranslation, t.data(), 3*sizeof(float), hipMemcpyHostToDevice);CUDA_CHECK;

    dim3 block = dim3(32,8,1);
    dim3 grid = dim3( (w + block.x -1) / block.x, (h+block.y -1) / block.y, 1);
    g_residualKernel <<<grid,block>>> (d_ptrGrayRef, d_ptrDepthRef, d_ptrGrayCur, d_ptrRotation,
                                d_ptrTranslation, fx, fy, cx, cy, w, h, d_residuals);
    hipDeviceSynchronize();


    hipFree(d_ptrRotation);CUDA_CHECK;
    hipFree(d_ptrTranslation);CUDA_CHECK;
    //hipFree(d_residuals);CUDA_CHECK;
    //hipUnbindTexture(texGrayCur);


}



__global__ void computeHuberWeightsKernel(float* weights, const float* residuals, int n, float k)
{
	//Compute index
	int x = threadIdx.x + blockDim.x * blockIdx.x;
	int y = threadIdx.y + blockDim.y * blockIdx.y;
	int z = threadIdx.z + blockDim.z * blockIdx.z;
	int i = x;
	//Do bounds check
	if(i<n && y < 1 && z < 1)
	{
		//compute robust Huber weights
        float w;
        if (std::abs(residuals[i]) <= k)
            w = 1.0f;
        else
            w = k / std::abs(residuals[i]);
        weights[i] = w;
	}
}



struct varianceshifteop
    : std::unary_function<float, float>
{
    varianceshifteop(float m)
        : mean(m)
    { /* no-op */ }

    const float mean;

    __device__ float operator()(float data) const
    {
    	return (data-mean)*(data-mean);
    }
};



void DVO::computeWeights(float* d_residuals, float* d_weights, int n)
{
#if 0
    // no weighting
    for (int i = 0; i < n; ++i)
        weights[i] = 1.0f;
#if 0
    // squared residuals
    for (int i = 0; i < n; ++i)
        residuals[i] = residuals[i] * residuals[i];
    return;
#endif
#endif

    float mean, stdDev;

    // wrap raw pointer with a device_ptr
    thrust::device_ptr<float> dp_residuals = thrust::device_pointer_cast(d_residuals);

    // sum elements and divide by the number of elements
    mean = thrust::reduce(
        dp_residuals,
        dp_residuals+n,
        0.0f,
        thrust::plus<float>()) / n;

    // shift elements by mean, square, and add them
    float variance = thrust::transform_reduce(
    		dp_residuals,
    		dp_residuals+n,
            varianceshifteop(mean),
            0.0f,
            thrust::plus<float>());

    // standard dev is just a sqrt away
    stdDev = std::sqrt(variance);

    float k = 1.345f * stdDev;

    dim3 block = dim3(512,1,1);
    dim3 grid = dim3((n+block.x-1) / block.x,
		1,
		1);
    computeHuberWeightsKernel<<<grid,block>>>(d_weights, d_residuals, n, k);
    hipDeviceSynchronize(); CUDA_CHECK;

}

__global__ void applyWeightsKernel(const float* weights, float* residuals, int n)
{
	//Compute index
	int x = threadIdx.x + blockDim.x * blockIdx.x;
	int y = threadIdx.y + blockDim.y * blockIdx.y;
	int z = threadIdx.z + blockDim.z * blockIdx.z;
	int i = x;
	//Do bounds check
	if(i<n && y < 1 && z < 1)
	{
		residuals[i] = residuals[i] * weights[i];
	}
}


void DVO::applyWeights(const float* d_weights, float* d_residuals, int n)
{

    dim3 block = dim3(512,1,1);
    dim3 grid = dim3((n+block.x-1) / block.x,
		1,
		1);
    applyWeightsKernel<<<grid,block>>>(d_weights, d_residuals, n);
    hipDeviceSynchronize(); CUDA_CHECK;

}


void DVO::deriveNumeric(const cv::Mat &grayRef, const cv::Mat &depthRef,
                                  const cv::Mat &grayCur, const cv::Mat &depthCur,
                                  const Eigen::VectorXf &xi, const Eigen::Matrix3f &K,
                                  float* residuals, float* J)
{
    float epsilon = 1e-6;
    float scale = 1.0f / epsilon;

    int w = grayRef.cols;
    int h = grayRef.rows;
    int n = w*h;

    // calculate per-pixel residuals
    //calculateError(convertToContGpuMat(grayRef), convertToContGpuMat(depthRef), convertToContGpuMat(grayCur), convertToContGpuMat(depthCur), xi, K, residuals);
    calculateError(grayRef, depthRef, grayCur, depthCur, xi, K, residuals);

    // create and fill Jacobian column by column
    float* residualsInc = new float[n];
    for (int j = 0; j < 6; ++j)
    {
        Eigen::VectorXf unitVec = Eigen::VectorXf::Zero(6);
        unitVec[j] = epsilon;

        // left-multiplicative increment on SE3
        Eigen::VectorXf xiEps = Sophus::SE3f::log(Sophus::SE3f::exp(unitVec) * Sophus::SE3f::exp(xi));

        //calculateError(convertToContGpuMat(grayRef), convertToContGpuMat(depthRef), convertToContGpuMat(grayCur), convertToContGpuMat(depthCur), xiEps, K, residualsInc);
        calculateError(grayRef, depthRef, grayCur, depthCur, xiEps, K, residualsInc);

        for (int i = 0; i < n; ++i)
            J[i*6 + j] = (residualsInc[i] - residuals[i]) * scale;
    }
    delete[] residualsInc;
}


__global__ void computeJtRIntermediateResultKernel(float* out, const float* J, const float* residuals, int m, int j)
{
	//Compute index
	int x = threadIdx.x + blockDim.x * blockIdx.x;
	int y = threadIdx.y + blockDim.y * blockIdx.y;
	int z = threadIdx.z + blockDim.z * blockIdx.z;
	int i = x;
	if(i<m && y < 1 && z < 1)
	{
		out[i] = J[i*6 + j] * residuals[i];
	}
}

void DVO::compute_JtR(float* d_J, const float* d_residuals, Vec6f &b, int validRows)
{

    int n = 6;
    int m = validRows;

    float alpha = 1;
    float beta = 0;

    float *d_y;
    hipMalloc(&d_y,sizeof(float)*6);

    hipblasSgemv(handle,HIPBLAS_OP_N,n,m,&alpha,d_J,n,d_residuals,1,&beta,d_y,1);

    float *res = new float[6];
    hipMemcpy(res,d_y,sizeof(float)*6,hipMemcpyDeviceToHost);

    for(int i = 0; i < 6; i++) {
        b[i] = res[i];
    }

    delete[] res;
    hipFree(d_y);

}

__global__ void JtJKernel(const float* d_J,  const float* d_weights, int validRows, bool useWeights, float *d_res) {

    int i = threadIdx.x + blockDim.x*blockIdx.x;
    int m = blockIdx.y;

    int n = 6;

    int k = floor( ( 2.0f*n+1 - sqrtf( (2.0f*n+1.0f)*(2.0f*n+1.0f) - 8.0f*m ) ) / 2.0f ) ;
    int j = k + (m - n*k + k*(k-1)/2) ;


    if(i < validRows) {

        float valSqr;
    	valSqr = d_J[i*6 + j] * d_J[i*6 + k];
    	if (useWeights)
    	   valSqr *= d_weights[i];

        d_res[i + m*validRows] = valSqr;
    }
}

__global__ void WJKernel(const float* d_J, const float* d_weights, int validRows, float *d_res) {
    int m = threadIdx.x + blockDim.x*blockIdx.x;
    int n = threadIdx.y;

    //extern __shared__ float s_weights[];

    /*if(m < validRows) {
        s_weights[m] = d_weights[m];
    }

    __syncthreads();
*/
    if(m < validRows) {
        d_res[n + m*6] = d_weights[m] * d_J[n + m*6];
    }
}


void DVO::compute_JtJ(const float* d_J, Mat6f &A, const float* d_weights, int validRows, bool useWeights)
{
    /*
    int n = 6;
    int m = validRows;

    dim3 block = dim3(128,1,1);
    // matrix A has 21 unique elements due to symmetry
	dim3 grid = dim3((m+block.x-1) / block.x,21,1);

    float *d_res;
    hipMalloc(&d_res, sizeof(float)*21*m);CUDA_CHECK;
    thrust::device_ptr<float> dp_res = thrust::device_pointer_cast(d_res);


    JtJKernel <<<grid,block>>>(d_J, d_weights, validRows, useWeights, d_res);

    hipStream_t stream0;
    hipStreamCreate ( &stream0) ;

    //float *res = new float[36];

    //hipMemcpy(res,d_res,sizeof(float)*36,hipMemcpyDeviceToHost);

    // column-major

    /*float *res = new float[21*m];
    hipMemcpy(res,d_res,sizeof(float)*21*m,hipMemcpyDeviceToHost);
    ///////
    hipDeviceSynchronize();

    for(int i = 0; i < 21; i++) {
        if(i % 2 == 0) {
            thrust::cuda::par.on(stream0);
        } else {
            thrust::cuda::par.on(0);
        }

        float val = thrust::reduce(
            dp_res + i*m,
            dp_res +i*m +m,
            0.0f,
            thrust::plus<float>());
        /*float val;
        for(int l = 0; l < m; l++) {
            val += res[i*m +l];
        }
        int k = floor( ( 2.0f*n+1 - sqrtf( (2.0f*n+1.0f)*(2.0f*n+1.0f) - 8.0f*i ) ) / 2.0f ) ;
        int j = k + (i - n*k + k*(k-1)/2) ;

        A(j,k) = val;
        A(k,j) = val;
    }
    //delete[] res;
    hipFree(d_res);
*/
/*
    int n = 6;
    int m = validRows;

    float *J = new float[n*m];
    hipMemcpy(J,d_J, sizeof(float)*n*m, hipMemcpyDeviceToHost);

    float *weights = new float[m];
    hipMemcpy(weights,d_weights, sizeof(float)*m, hipMemcpyDeviceToHost);

    // compute A = Jt*J
    for (int k = 0; k < n; ++k)
    {
        for (int j = k; j < n; ++j)
        {
            float val = 0.0f;
            for (int i = 0; i < m; ++i)
            {
                float valSqr = J[i*6 + j] * J[i*6 + k];
                if (useWeights)
                    valSqr *= weights[i];
                val += valSqr;
            }
            A(k, j) = val;
            A(j,k) = val;
        }
    }

    delete[] J;
    delete[] weights;

//    t.end();


//    std::cout << "CPU: " << t.get() << std::endl;
*/
    int n = 6;
    int m = validRows;

    float *d_res;
    hipMalloc(&d_res,sizeof(float)*36);

    float alpha = 1;
    float beta = 0;


    if(useWeights) {
        float *d_WJ;
        hipMalloc(&d_WJ,sizeof(float)*n*m);

        dim3 block = dim3(32,6,1);
        dim3 grid = dim3( (m + block.x -1) / block.x, 1, 1);

        WJKernel <<<grid, block>>> (d_J, d_weights, validRows,d_WJ);
        //hipDeviceSynchronize(); CUDA_CHECK;

        //hipblasSdgmm(handle, HIPBLAS_SIDE_RIGHT, n,m, d_J, n, d_weights, 0, d_WJ, n);
        hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, n, n, m, &alpha, d_WJ, n, d_J, n, &beta, d_res, n);
        hipFree(d_WJ);

    } else {
        hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, n, n, m, &alpha, d_J, n, d_J, n, &beta, d_res, n);
    }

    // column-major
    float *res = new float[36];
    hipMemcpy(res,d_res,sizeof(float)*36,hipMemcpyDeviceToHost);

    // column-major
    for(int k = 0; k < n; k++) {
        for(int j = 0; j < n; j++) {
            A(j,k) = res[k + 6*j];
        }
    }

    delete[] res;
    hipFree(d_res);


}

__device__ void rotateAndTranslate(float *rot,float *t, float *v, float *res)
{
	for(int i = 0;i<3;i++)
	{
		float sum = 0.f;
		for(int j = 0;j<3;j++)
		{
			sum += rot[i+3*j]*v[j];
		}
		res[i] = sum + t[i];
	}

}

__device__ void multiply(float *mat,float *v,float *res)
{
	for(int i = 0;i<3;i++)
	{
		float sum = 0.f;
		for(int j = 0;j<3;j++)
		{
			sum += mat[i+3*j]*v[j];
		}
		res[i] = sum;
	}

}


__global__ void computeAnalyticalGradient(float *d_K,float* d_ptrGrayRef,float* d_ptrDepthRef,float* d_ptrGrayCur,float * d_rotMat, float* d_t,
		float *d_gradx,float *d_grady,int w, int h,float *d_J,float *d_residuals)
{

	int x = threadIdx.x + blockDim.x*blockIdx.x;
	int y = threadIdx.y + blockDim.y*blockIdx.y;


	if(x<w && y<h)
	{
		size_t idx = x + (size_t)w*y;

		float residual = 0.0f;

		float fx = d_K[0];
		float fy = d_K[4];
		float cx = d_K[6];
		float cy = d_K[7];
		float fxInv = 1.0f / fx;
    	float fyInv = 1.0f / fy;

		bool innerIfExecuted=false;

        // project 2d point back into 3d using its depth
        float dRef = d_ptrDepthRef[idx];
        if (dRef > 0.0f)
        {
            float x0 = (static_cast<float>(x) - cx) * fxInv;
            float y0 = (static_cast<float>(y) - cy) * fyInv;
            float scale = 1.0f;
            //scale = std::sqrt(x0*x0 + y0*y0 + 1.0);
            dRef = dRef * scale;
            x0 = x0 * dRef;
            y0 = y0 * dRef;

            // transform reference 3d point into current frame
            // reference 3d point
            // Eigen::Vector3f pt3Ref(x0, y0, dRef);
            float pt3Ref[3] = {x0,y0,dRef};
            float pt3[3];

            rotateAndTranslate(d_rotMat,d_t,pt3Ref,pt3);

            if (pt3[2] > 0.0f)
            {
            	 /*
                // project 3d point to 2d
                float pt2CurH[3];
                multiply(d_K,pt3,pt2CurH);
            	//Eigen::Vector3f pt2CurH = K * pt3;

                float ptZinv = 1.0f / pt2CurH[2];
                float px = pt2CurH[0] * ptZinv;
                float py = pt2CurH[1] * ptZinv;
            	*/

            	float px = (fx * pt3[0] + cx * pt3[2]) / pt3[2];
            	float py = (fy * pt3[1] + cy * pt3[2]) / pt3[2];

                // Computing error

                float valCur = d_interpolate(d_ptrGrayCur, px, py, w, h);
			    if (!isnan(valCur))
			    {
				   float valRef = d_ptrGrayRef[idx];
				   float valDiff = valRef - valCur;
				   residual = valDiff;
			    }

                // compute interpolated image gradient
                float dX = d_interpolate(d_gradx, px, py, w, h);
                float dY = d_interpolate(d_grady, px, py, w, h);

                if (!isnan(dX) && !isnan(dY))
                {
                	innerIfExecuted = true;
                    dX = fx * dX;
                    dY = fy * dY;
                    float pt3Zinv = 1.0f / pt3[2];

                    // shorter computation
                   		d_J[idx*6 + 0] = -1.0f*dX * pt3Zinv;
		                d_J[idx*6 + 1] = -1.0f*dY * pt3Zinv;
		                d_J[idx*6 + 2] = (dX * pt3[0] + dY * pt3[1]) * pt3Zinv * pt3Zinv;
		                d_J[idx*6 + 3] = (dX * pt3[0] * pt3[1]) * pt3Zinv * pt3Zinv + dY * (1 + (pt3[1] * pt3Zinv) * (pt3[1] * pt3Zinv));
		                d_J[idx*6 + 4] = - dX * (1.0 + (pt3[0] * pt3Zinv) * (pt3[0] * pt3Zinv)) - (dY * pt3[0] * pt3[1]) * pt3Zinv * pt3Zinv;
		                d_J[idx*6 + 5] = -1.0f*(- dX * pt3[1] + dY * pt3[0]) * pt3Zinv;
                }
            }
        }

        if(!innerIfExecuted) {
        	for (int j=0;j<6;j++) {

        			d_J[idx*6 + j] =  0.0f;

        	}
        }

        d_residuals[idx] = residual;

    }
}



void DVO::deriveAnalytic(const cv::gpu::GpuMat &grayRef, const cv::gpu::GpuMat &depthRef,
                   const cv::gpu::GpuMat &grayCur, const cv::gpu::GpuMat &depthCur,
                   const cv::gpu::GpuMat &gradX, const cv::gpu::GpuMat &gradY,
                   const Eigen::VectorXf &xi, const Eigen::Matrix3f &K,
                   float* d_residuals, float* d_J)
{
    // reference input images
    int w = grayRef.cols;
    int h = grayRef.rows;
    int n = w*h;

    // convert SE3 to rotation matrix and translation vector
    Eigen::Matrix3f rotMat;
    Eigen::Vector3f t;
    convertSE3ToTf(xi, rotMat, t);


    //calculateError(grayRef, depthRef, grayCur, depthCur, xi, K, d_residuals);
    // Using multi threading
    dim3 block =  dim3(32,32,1);
    dim3 grid = dim3((w+block.x-1)/block.x,(h+block.y-1)/block.y,1);

    // Allocating device memory
    float *d_gradx,*d_grady,*d_t,*d_K,*d_rotMat;

    hipMalloc(&d_rotMat,9*sizeof(float));CUDA_CHECK;
    hipMalloc(&d_K,9*sizeof(float));CUDA_CHECK;
    hipMalloc(&d_t,3*sizeof(float));CUDA_CHECK;

    d_gradx = (float*) gradX.data;
    d_grady = (float*) gradY.data;

    float* d_ptrGrayRef = (float*)grayRef.ptr();
	float* d_ptrDepthRef = (float*)depthRef.ptr();
	float* d_ptrGrayCur = (float*)grayCur.ptr();
	float* d_ptrDepthCur = (float*)depthCur.ptr();

    hipMemcpy(d_rotMat,rotMat.data(),9*sizeof(float),hipMemcpyHostToDevice);CUDA_CHECK;
    hipMemcpy(d_K,K.data(),9*sizeof(float),hipMemcpyHostToDevice);CUDA_CHECK;
    hipMemcpy(d_t,t.data(),3*sizeof(float),hipMemcpyHostToDevice);CUDA_CHECK;

    computeAnalyticalGradient<<<grid,block>>>(d_K,d_ptrGrayRef,d_ptrDepthRef,d_ptrGrayCur, d_rotMat,d_t,d_gradx,d_grady,w,h,d_J,d_residuals);
    hipDeviceSynchronize();

    hipFree(d_K);CUDA_CHECK;
    hipFree(d_rotMat);CUDA_CHECK;
    hipFree(d_t);CUDA_CHECK;

}

cv::gpu::GpuMat DVO::convertToContGpuMat(const cv::Mat &m) {
    cv::gpu::GpuMat gpuM = cv::gpu::createContinuous(m.rows, m.cols, m.type());
    gpuM.upload(m);
    return gpuM;
}

void DVO::buildPyramid(const cv::Mat &depth, const cv::Mat &gray, std::vector<cv::gpu::GpuMat> &depthPyramid, std::vector<cv::gpu::GpuMat> &grayPyramid)
{
    grayPyramid.push_back(convertToContGpuMat(gray));
    depthPyramid.push_back(convertToContGpuMat(depth));

    for (int i = 1; i < numPyramidLevels_; ++i)
    {
        // downsample grayscale image
        cv::gpu::GpuMat grayDown = downsampleGray(grayPyramid[i-1]);
        grayPyramid.push_back(grayDown);

        // downsample depth image
        cv::gpu::GpuMat depthDown = downsampleDepth(depthPyramid[i-1]);
        depthPyramid.push_back(depthDown);

    }



}


void DVO::align(const cv::Mat &depthRef, const cv::Mat &grayRef, const cv::Mat &depthCur, const cv::Mat &grayCur, Eigen::Matrix4f &pose)
{
    // downsampling

    std::vector<cv::gpu::GpuMat> grayRefGPUPyramid;
    std::vector<cv::gpu::GpuMat> depthRefGPUPyramid;

    buildPyramid(depthRef, grayRef, depthRefGPUPyramid, grayRefGPUPyramid);

    std::vector<cv::gpu::GpuMat> grayCurGPUPyramid;
    std::vector<cv::gpu::GpuMat> depthCurGPUPyramid;

    buildPyramid(depthCur, grayCur, depthCurGPUPyramid, grayCurGPUPyramid);

    align(depthRefGPUPyramid, grayRefGPUPyramid, depthCurGPUPyramid, grayCurGPUPyramid, pose);
}


void DVO::align(const std::vector<cv::gpu::GpuMat> &depthRefGPUPyramid, const std::vector<cv::gpu::GpuMat> &grayRefGPUPyramid,
                const std::vector<cv::gpu::GpuMat> &depthCurGPUPyramid, const std::vector<cv::gpu::GpuMat> &grayCurGPUPyramid,
                Eigen::Matrix4f &pose)
{

    Vec6f xi;
    convertTfToSE3(pose, xi);

    Vec6f lastXi = Vec6f::Zero();

    int maxLevel = numPyramidLevels_-1;
    int minLevel = 1;
    float initGradDescStepSize = 1e-3f;
    float gradDescStepSize = initGradDescStepSize;

    Mat6f A;
    Mat6f diagMatA = Mat6f::Identity();
    Vec6f delta;

    for (int lvl = maxLevel; lvl >= minLevel; --lvl)
    {
        float lambda = 0.1f;

        int w = sizePyramid_[lvl].width;
        int h = sizePyramid_[lvl].height;
        int n = w*h;

        cv::gpu::GpuMat grayRef = grayRefGPUPyramid[lvl];
        cv::gpu::GpuMat depthRef = depthRefGPUPyramid[lvl];
        cv::gpu::GpuMat grayCur = grayCurGPUPyramid[lvl];
        cv::gpu::GpuMat depthCur = depthCurGPUPyramid[lvl];
        Eigen::Matrix3f kLevel = kPyramid_[lvl];
        //std::cout << "level " << level << " (size " << depthRef.cols << "x" << depthRef.rows << ")" << std::endl;

        // compute gradient images
        computeGradient(grayCur, gradX_[lvl], 0);
        computeGradient(grayCur, gradY_[lvl], 1);

        float errorLast = std::numeric_limits<float>::max();
        for (int itr = 0; itr < numIterations_; ++itr)
        {
            // compute residuals and Jacobian
#if 0
            deriveNumeric(grayRef, depthRef, grayCur, depthCur, xi, kLevel, residuals_[lvl], J_[lvl]);
#else
            deriveAnalytic(grayRef, depthRef, grayCur, depthCur, gradX_[lvl], gradY_[lvl], xi, kLevel, d_residuals_[lvl], d_J_[lvl]);
#endif

#if 0
            // compute and show error image
            cv::Mat errorImage;
            calculateErrorImage(residuals_[lvl], grayRef.cols, grayRef.rows, errorImage);
            std::stringstream ss;
            ss << "residuals_" << lvl << "_";
            ss << std::setw(2) << std::setfill('0') << itr << ".png";
            cv::imwrite(ss.str(), errorImage);
            cv::imshow("error", errorImage);
            cv::waitKey(0);
#endif

            // calculate error
            float error = calculateError(d_residuals_[lvl], n);
            if (useWeights_)
            {
                // compute robust weights
                computeWeights(d_residuals_[lvl], d_weights_[lvl], n);
                // apply robust weights
                applyWeights(d_weights_[lvl], d_residuals_[lvl], n);
            }

            // compute update
            Vec6f b;
            compute_JtR(d_J_[lvl], d_residuals_[lvl], b, n);

            if (algo_ == GradientDescent)
            {
                // Gradient Descent
                delta = -gradDescStepSize * b * (1.0f / b.norm());
            }
            else if (algo_ == GaussNewton)
            {
                // Gauss-Newton algorithm
                compute_JtJ(d_J_[lvl], A, d_weights_[lvl], n, useWeights_);
                // solve using Cholesky LDLT decomposition
                delta = -(A.ldlt().solve(b));
            }
            else if (algo_ == LevenbergMarquardt)
            {
                // Levenberg-Marquardt algorithm
                compute_JtJ(d_J_[lvl], A, d_weights_[lvl], n, useWeights_);
                diagMatA.diagonal() = lambda * A.diagonal();
                delta = -((A + diagMatA).ldlt().solve(b));
            }

            // apply update: left-multiplicative increment on SE3
            lastXi = xi;
            xi = Sophus::SE3f::log(Sophus::SE3f::exp(delta) * Sophus::SE3f::exp(xi));
#if 0
            std::cout << "delta = " << delta.transpose() << " size = " << delta.rows() << " x " << delta.cols() << std::endl;
            std::cout << "xi = " << xi.transpose() << std::endl;
#endif

            // compute error again
            error = calculateError(d_residuals_[lvl], n);

            if (algo_ == LevenbergMarquardt)
            {
                if (error >= errorLast)
                {
                    lambda = lambda * 5.0f;
                    xi = lastXi;

                    if (lambda > 5.0f)
                        break;
                }
                else
                {
                    lambda = lambda / 1.5f;
                }
            }
            else if (algo_ == GaussNewton)
            {
                // break if no improvement (0.99 or 0.995)
                if (error / errorLast > 0.995f)
                    break;
            }
            else if (algo_ == GradientDescent)
            {
                if (error >= errorLast)
                {
                    gradDescStepSize = gradDescStepSize * 0.5f;
                    if (gradDescStepSize <= initGradDescStepSize * 0.01f)
                        gradDescStepSize = initGradDescStepSize * 0.01f;
                    xi = lastXi;
                }
                else
                {
                    gradDescStepSize = gradDescStepSize * 2.0f;
                    if (gradDescStepSize >= initGradDescStepSize * 100.0f)
                        gradDescStepSize = initGradDescStepSize * 100.0f;

                    // break if no improvement (0.99 or 0.995)
                    if (error / errorLast > 0.995f)
                        break;
                }
            }

            errorLast = error;
        }
    }

    // store to output pose
    convertSE3ToTf(xi, pose);
}
